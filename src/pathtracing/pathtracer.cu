#include "hip/hip_runtime.h"
#include "pathtracer.cuh"

#include <vector>
#include <iostream>
#include <thrust/random.h>
#include <globals.h>


PathTracer::PathTracer(int width, int height) :
    imageWidth(width), imageHeight(height) {
}

struct Camera {
	int x;

	/*__device__ void dos() {
		x++;
	}*/
};

__global__
void testCameraKernel(Camera* camera) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //int stride = blockDim.x * gridDim.x;

	if (index == 0) {
		//camera->dos();
		printf("%i", camera->x);
	}

    //camera->w = 5;
    
    //for(int i = index; i < n; i += stride) {
    //    x[3*i] = 200.0f;
    //    x[3*i+1] = 200.0f;
    //    x[3*i+2] = 200.0f; 
    //}
}

__global__ void doNothingKernel() {

}

std::vector<float> PathTracer::getFrameBuffer() {
	//doNothingKernel << <1, 1 >> > ();
	//return std::vector<float>();

	Camera * testCamera = new Camera();
	hipMallocManaged(&testCamera, sizeof(Camera));
	testCameraKernel << <1, 1 >> > (testCamera);
	return std::vector<float>();


	//testCamera->x = 5;

	//return;

	//Scene *scene = new Scene();
    //Camera *camera = new Camera();

    //int numPixels = imageWidth * imageHeight;
    //int dataSize = numPixels * 3;

    //float *x;
    //hipMallocManaged(&x, dataSize * sizeof(float));
    //hipMallocManaged(&camera, sizeof(camera));

    //int blockSize = 256;
    //int numBlocks = (numPixels + blockSize - 1) / blockSize;
    //test<<<numBlocks,blockSize>>>();

	//Camera *camera = new Camera();
	//hipMallocManaged(&camera, sizeof(Camera));

	//camera->x = 6;
	//std::cout << camera->x << std::endl;
	
	//test<< <numBlocks, blockSize >> > (camera);

    //hipDeviceSynchronize();

	

    //std::vector<float> result = std::vector<float>(x, x + dataSize);

    //hipFree(x);

    //std::cout << "done" << std::endl;
    //std::cout << camera->w << std::endl;

    //std::vector<float> result = std::vector<float>(3 * imageWidth * imageHeight);
    //return std::vector<float>();
}
