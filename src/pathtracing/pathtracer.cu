#include "hip/hip_runtime.h"
#include "pathtracer.cuh"

#include <vector>
#include <iostream>
#include <thrust/random.h>
#include <globals.h>


PathTracer::PathTracer(int width, int height) :
    imageWidth(width), imageHeight(height) {
}

struct Camera {
	int x;

	/*__device__ void dos() {
		x++;
	}*/
};

__global__
void testCameraKernel(Camera* camera) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //int stride = blockDim.x * gridDim.x;

	if (index == 0) {
		//camera->dos();
		printf("%i", camera->x);
	}

    //camera->w = 5;
    
    //for(int i = index; i < n; i += stride) {
    //    x[3*i] = 200.0f;
    //    x[3*i+1] = 200.0f;
    //    x[3*i+2] = 200.0f; 
    //}
}

__global__ void doNothingKernel() {

}

__global__ void fillPBO(int n, uchar4 *pbo, float frame) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		pbo[index].w = 0;
		pbo[index].x = 0;
		pbo[index].y = 0;
		pbo[index].z = (int)frame;
	}
}

void PathTracer::pathtrace(void *pbo, float frame) {
	//std::cout << frame << std::endl;
	int blockSize = 256;
	int numBlocks = (imageWidth * imageHeight + blockSize - 1) / blockSize;

	fillPBO<<<numBlocks,blockSize>>>(imageWidth * imageHeight, (uchar4*)pbo, frame);
	hipDeviceSynchronize();

	return;


	//doNothingKernel << <1, 1 >> > ();
	//return std::vector<float>();

	//Camera * testCamera = new Camera();
	//hipMallocManaged(&testCamera, sizeof(Camera));
	//testCameraKernel << <1, 1 >> > (testCamera);
	//return std::vector<float>();
	//return;

	//testCamera->x = 5;

	//return;

	//Scene *scene = new Scene();
    //Camera *camera = new Camera();

    //int numPixels = imageWidth * imageHeight;
    //int dataSize = numPixels * 3;

    //float *x;
    //hipMallocManaged(&x, dataSize * sizeof(float));
    //hipMallocManaged(&camera, sizeof(camera));

    //int blockSize = 256;
    //int numBlocks = (numPixels + blockSize - 1) / blockSize;
    //test<<<numBlocks,blockSize>>>();

	//Camera *camera = new Camera();
	//hipMallocManaged(&camera, sizeof(Camera));

	//camera->x = 6;
	//std::cout << camera->x << std::endl;
	
	//test<< <numBlocks, blockSize >> > (camera);

    //hipDeviceSynchronize();

	

    //std::vector<float> result = std::vector<float>(x, x + dataSize);

    //hipFree(x);

    //std::cout << "done" << std::endl;
    //std::cout << camera->w << std::endl;

    //std::vector<float> result = std::vector<float>(3 * imageWidth * imageHeight);
    //return std::vector<float>();
}
