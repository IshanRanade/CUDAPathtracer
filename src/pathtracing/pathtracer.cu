#include "hip/hip_runtime.h"
#include "pathtracer.cuh"

#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include "scene/camera.cuh"
#include "scene/scene.cuh"

PathTracer::PathTracer(int width, int height) :
    imageWidth(width), imageHeight(height) {
}

__global__
void test(int n, float *x, Camera *camera) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    //camera->w = 5;
    
    for(int i = index; i < n; i += stride) {
        x[3*i] = 200.0f;
        x[3*i+1] = 200.0f;
        x[3*i+2] = 200.0f;
    }
}

std::vector<float> PathTracer::getFrameBuffer() {
    Scene *scene = new Scene();
    Camera *camera = new Camera();

    int numPixels = imageWidth * imageHeight;
    int dataSize = numPixels * 3;

    float *x;
    hipMallocManaged(&x, dataSize * sizeof(float));
    hipMallocManaged(&camera, sizeof(camera));

    int blockSize = 256;
    int numBlocks = (numPixels + blockSize - 1) / blockSize;
    test<<<numBlocks,blockSize>>>(numPixels, x, camera);

    hipDeviceSynchronize();

    std::vector<float> result = std::vector<float>(x, x + dataSize);

    hipFree(x);

    std::cout << "done" << std::endl;
    //std::cout << camera->w << std::endl;

    //std::vector<float> result = std::vector<float>(3 * imageWidth * imageHeight);
    return result;
}
