#include "hip/hip_runtime.h"
#include "camera.cuh"

Camera::Camera():
    Camera(400, 400)
{
    look = Vector3f(0,0,-1);
    up = Vector3f(0,1,0);
    right = Vector3f(1,0,0);
}

Camera::Camera(unsigned int w, unsigned int h):
    Camera(w, h, Vector3f(0,0,10), Vector3f(0,0,0), Vector3f(0,1,0))
{}

Camera::Camera(unsigned int w, unsigned int h, const Vector3f &e, const Vector3f &r, const Vector3f &worldUp):
    fovy(45),
    width(w),
    height(h),
    nearClip(0.1f),
    farClip(1000),
    eye(e),
    ref(r),
    world_up(worldUp)
{
    RecomputeAttributes();
}

Camera::Camera(const Camera &c):
    fovy(c.fovy),
    width(c.width),
    height(c.height),
    nearClip(c.nearClip),
    farClip(c.farClip),
    aspect(c.aspect),
    eye(c.eye),
    ref(c.ref),
    look(c.look),
    up(c.up),
    right(c.right),
    world_up(c.world_up),
    V(c.V),
    H(c.H)
{}

void Camera::CopyAttributes(const Camera &c)
{
    fovy = c.fovy;
    nearClip = c.nearClip;
    farClip = c.farClip;
    eye = c.eye;
    ref = c.ref;
    look = c.look;
    up = c.up;
    right = c.right;
    width = c.width;
    height = c.height;
    aspect = c.aspect;
    V = c.V;
    H = c.H;
}

void Camera::RecomputeAttributes()
{
    look = glm::normalize(ref - eye);
    right = glm::normalize(glm::cross(look, world_up));
    up = glm::cross(right, look);

    float tan_fovy = tan(glm::radians(fovy/2));
    float len = glm::length(ref - eye);
    aspect = width/(float)height;
    V = up*len*tan_fovy;
    H = right*len*aspect*tan_fovy;
}

glm::mat4 Camera::GetViewProj() const
{
    return glm::perspective(glm::radians(fovy), width / (float)height, nearClip, farClip) * glm::lookAt(eye, ref, up);
}

void Camera::RotateAboutUp(float deg)
{
    deg = glm::radians(deg);
    glm::mat4 rotation = glm::rotate(glm::mat4(1.0f), deg, up);
    ref = ref - eye;
    ref = glm::vec3(rotation * glm::vec4(ref, 1));
    ref = ref + eye;
    RecomputeAttributes();
}
void Camera::RotateAboutRight(float deg)
{
    deg = glm::radians(deg);
    glm::mat4 rotation = glm::rotate(glm::mat4(1.0f), deg, right);
    ref = ref - eye;
    ref = glm::vec3(rotation * glm::vec4(ref, 1));
    ref = ref + eye;
    RecomputeAttributes();
}

void Camera::TranslateAlongLook(float amt)
{
    glm::vec3 translation = look * amt;
    eye += translation;
    ref += translation;
}

void Camera::TranslateAlongRight(float amt)
{
    glm::vec3 translation = right * amt;
    eye += translation;
    ref += translation;
}
void Camera::TranslateAlongUp(float amt)
{
    glm::vec3 translation = up * amt;
    eye += translation;
    ref += translation;
}

Ray Camera::Raycast(const Point2f &pt) const
{
    return Raycast(pt.x, pt.y);
}

Ray Camera::Raycast(float x, float y) const
{
    float ndc_x = (2.f*x/width - 1);
    float ndc_y = (1 - 2.f*y/height);
    return RaycastNDC(ndc_x, ndc_y);
}

Ray Camera::RaycastNDC(float ndc_x, float ndc_y) const
{
    glm::vec3 P = ref + ndc_x*H + ndc_y*V;
    Ray result(eye, glm::normalize(P - eye));
    return result;
}