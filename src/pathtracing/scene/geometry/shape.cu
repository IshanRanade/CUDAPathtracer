#include "shape.cuh"


void Shape::InitializeIntersection(Intersection *isect, float t, Point3f pLocal) const
{
    isect->point = Point3f(transform.T() * glm::vec4(pLocal, 1));
    ComputeTBN(pLocal, &(isect->normalGeometric), &(isect->tangent), &(isect->bitangent));
    //isect->uv = GetUVCoordinates(pLocal);
    isect->t = t;
}
