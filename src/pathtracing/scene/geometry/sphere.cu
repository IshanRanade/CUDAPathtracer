#include "hip/hip_runtime.h"
#include "sphere.cuh"


bool Sphere::Intersect(const Ray &ray, Intersection *isect) const {
    //Transform the ray
    Ray r_loc = ray.GetTransformedCopy(transform.invT());

    float A = pow(r_loc.direction.x, 2.f) + pow(r_loc.direction.y, 2.f) + pow(r_loc.direction.z, 2.f);
    float B = 2*(r_loc.direction.x*r_loc.origin.x + r_loc.direction.y * r_loc.origin.y + r_loc.direction.z * r_loc.origin.z);
    float C = pow(r_loc.origin.x, 2.f) + pow(r_loc.origin.y, 2.f) + pow(r_loc.origin.z, 2.f) - 1.f;//Radius is 1.f
    float discriminant = B*B - 4*A*C;
    //If the discriminant is negative, then there is no real root
    if(discriminant < 0){
        return false;
    }
    float t = (-B - sqrt(discriminant))/(2*A);
    if(t < 0)
    {
        t = (-B + sqrt(discriminant))/(2*A);
    }
    if(t >= 0)
    {
        Point3f P = glm::vec3(r_loc.origin + t*r_loc.direction);
        InitializeIntersection(isect, t, P);
        return true;
    }
    return false;
}

void Sphere::ComputeTBN(const Point3f& P, Normal3f* nor, Vector3f* tan, Vector3f* bit) const
{
    *nor = glm::normalize(transform.invTransT() * glm::normalize(P));
    //TODO: Compute tangent and bitangent
    *tan = glm::normalize(transform.T3() * glm::cross(Vector3f(0,1,0), (glm::normalize(P))));
    *bit = glm::normalize(glm::cross(*nor, *tan));
}

float Sphere::Area() const {
    return 0;
}